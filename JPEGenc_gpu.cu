#include"JPEGenc.h"

float JPEGencoder::encode_gpu() {
	hipEvent_t startTime, endTime;
	hipEventCreate(&startTime);
	hipEventCreate(&endTime);
	hipEventRecord(startTime);
	
	int r=8; // each block has 2^r threads
	int matrixSize=n*m;
	kernel_cntsm<<<((matrixSize+(1<<r)-1)>>r),(1<<r)>>>(mat_d,result_d,n,m);
	
	hipEventRecord(endTime);
	hipEventSynchronize(endTime);
	float duration=0;
	hipEventElapsedTime(&duration,startTime,endTime);
	return duration;
}